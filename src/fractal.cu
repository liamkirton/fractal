#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdint>
#include <iostream>
#include <memory>
#include <vector>

#include "hip/hip_runtime.h"
#include ""

#include "fractal.h"
#include "fixed_point.h"

constexpr uint64_t groups = 128;
constexpr uint64_t threads = 128;

constexpr uint32_t escape_block = 512;
constexpr uint32_t escape_limit = 512;

__global__ void mandelbrot_kernel(uint64_t *chunk_buffer, const uint64_t image_width, const uint64_t image_height, const double image_re, const double image_im, const double image_scale, const uint64_t image_chunk, const uint32_t escape_i) {
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int pixel_x = (image_chunk + tid) % image_width;
    const int pixel_y = (image_chunk + tid) / image_width;

    //const double re_c = image_re + (-2.0 + pixel_x * 3.0 / image_width) / image_scale;
    //const double im_c = image_im + (1.0 - pixel_y * 2.0 / image_height) / image_scale;

    fixed_point<1, 2> re_c(-2.0 + pixel_x * 3.0 / image_width);
    fixed_point<1, 2> im_c(1.0 - pixel_y * 2.0 / image_height);

    uint32_t escape = static_cast<uint32_t>(chunk_buffer[tid * 3]);
    /*double re_z = reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 1];
    double im_z = reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 2];
    double abs_z = 0.0;*/

    fixed_point<1, 2> re_z(reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 1]);
    fixed_point<1, 2> im_z(reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 1]);
    fixed_point<1, 2> abs_z;

    if (escape_i == 0) {
        escape = escape_limit;
        re_z.set(re_c);
        im_z.set(im_c);
    }

    if (escape == escape_limit) {
        for (uint32_t i = 0; i < escape_block; ++i) {
            /*double re_z_i = re_z;
            re_z = (re_z * re_z) - (im_z * im_z) + re_c;
            im_z = (2.0 * re_z_i * im_z) + im_c;
            abs_z = re_z * re_z + im_z * im_z;
            if (abs_z > 4.0) {
                escape = i + escape_i * escape_block;
                break;
            }*/
            fixed_point<1, 2> re_prod(re_z);
            fixed_point<1, 2> im_prod(im_z);

            re_prod.multiply(re_z);
            im_prod.multiply(im_z);

            fixed_point<1, 2> re_imed(im_prod);
            re_imed.negate();
            re_imed.add(re_prod);
            re_imed.add(re_c);

            fixed_point<1, 2> im_imed(2);
            im_imed.multiply(re_z);
            im_imed.multiply(im_z);
            im_imed.add(im_c);

            re_z.set(re_imed);
            im_z.set(im_imed);

            re_prod.set(re_z);
            re_prod.multiply(re_z);

            im_prod.set(im_z);
            im_prod.multiply(im_z);

            fixed_point<1, 2> abs(re_prod);
            abs.add(im_prod);

            if (abs.get_integer() > 4) {
                escape = i + escape_i * escape_block;
                break;
            }
        }
    }

    chunk_buffer[tid * 3] = escape;
    reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 1] = 0;// re_z;
    reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 2] = 0;// im_z;
}

__global__ void mandelbrot_kernel_colour(uint64_t *chunk_buffer, uint32_t *image_chunk_buffer, const uint64_t image_width, const uint64_t image_height, const uint64_t image_chunk) {
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

    uint32_t escape = chunk_buffer[tid * 3];
    double re_z = reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 1];
    double im_z = reinterpret_cast<double *>(chunk_buffer)[tid * 3 + 2];
    double abs_z = sqrtf(re_z * re_z + im_z * im_z);

    //double hue = 360.0 * log(1.0 * escape) / log(1.0 * escape_limit) + 1.0 - (log(log(abs_z)) / log(2.0)); 
    double hue = 360.0 * (log(1.0 * escape) - log(log(abs_z))) / (log(1.0 * escape_limit) + log(2.0));
    double sat = 0.85;
    double val = 1.0;

    hue = fmod(hue, 360.0);
    hue /= 60;

    double hue_fract = hue - floor(hue);
    double p = val * (1.0 - sat);
    double q = val * (1.0 - sat * hue_fract);
    double t = val * (1.0 - sat * (1.0 - hue_fract));

    double r = 0;
    double g = 0;
    double b = 0;

    if (escape < escape_limit) {
        switch ((static_cast<unsigned char>(floor(hue)) + 3) % 6) {
        case 0:
            r = val; g = t; b = p;
            break;
        case 1:
            r = q; g = val; b = p;
            break;
        case 2:
            r = p; g = val; b = t;
            break;
        case 3:
            r = p; g = q; b = val;
            break;
        case 4:
            r = t; g = p; b = val;
            break;
        case 5:
            r = val; g = p; b = q;
            break;
        default:
            break;
        }
        r = floor(r * 255); g = floor(g * 255); b = floor(b * 255);
    }

    image_chunk_buffer[tid] =
        (static_cast<unsigned char>(r)) |
        (static_cast<unsigned char>(g) << 8) |
        (static_cast<unsigned char>(b) << 16) |
        (255 << 24);
}

uint64_t *chunk_buffer{ nullptr };
uint32_t *chunk_buffer_image{ nullptr };

int init(uint64_t image_width, uint64_t image_height) {
    hipError_t cuda_status;
    if ((cuda_status = hipSetDevice(0)) != hipSuccess) {
        std::wcout << L"ERROR: hipSetDevice() Failed. [" << cuda_status << L"]" << std::endl << std::endl;
        return -1;
    }

    hipMalloc(&chunk_buffer, groups * threads * 3 * sizeof(uint64_t));
    hipMalloc(&chunk_buffer_image, groups * threads * sizeof(uint32_t));

    return 0;
}

int uninit(uint64_t image_width, uint64_t image_height) {
    hipFree(chunk_buffer);
    hipFree(chunk_buffer_image);
    chunk_buffer = nullptr;
    chunk_buffer_image = nullptr;

    hipError_t cuda_status;
    if ((cuda_status = hipDeviceReset()) != hipSuccess) {
        std::wcout << L"ERROR: hipDeviceReset() Failed. [" << cuda_status << L"]" << std::endl << std::endl;
        return -1;
    }

    return 0;
}

int mandelbrot(uint32_t *image, const uint64_t image_width, const uint64_t image_height, const double image_center_re, const double image_center_im, const double image_scale) {
    hipError_t hipError_t;

    std::wcout << "[+] Image: z = " << image_center_re << " + " << image_center_im << "i; scale = " << (1.0 / image_scale) << "; "
        << (image_center_re + (-2.0 / image_width) / image_scale) << " : "
        << (image_center_im + (1.0 / image_height) / image_scale) << std::endl;

    std::wcout << L"[+] Chunks: "
        << 1 + image_width * image_height / (groups * threads)
        << L" " << std::flush;

    for (uint64_t image_chunk = 0; image_chunk < (image_width * image_height); image_chunk += (groups * threads)) {
        uint64_t chunk_size = std::min((image_width * image_height) - image_chunk, groups * threads);
        uint64_t chunk_groups = chunk_size / threads;
        hipMemset(chunk_buffer, 0, groups * threads * 5 * sizeof(uint64_t));
        hipMemset(chunk_buffer_image, 0, groups * threads * sizeof(uint32_t));

        std::wcout << L"+" << std::flush;

        for (uint32_t i = 0; i < (escape_limit / escape_block); ++i) {
            mandelbrot_kernel<<<static_cast<uint32_t>(chunk_groups), static_cast<uint32_t>(threads)>>>(chunk_buffer, image_width, image_height, image_center_re, image_center_im, image_scale, image_chunk, i);
            if ((hipError_t = hipDeviceSynchronize()) != hipSuccess) {
                std::wcout << std::endl << "[!] hipDeviceSynchronize(): hipError_t: " << hipError_t << std::endl;
                return -1;
            }
        }

        mandelbrot_kernel_colour<<<static_cast<uint32_t>(chunk_groups), static_cast<uint32_t>(threads)>>>(chunk_buffer, chunk_buffer_image, image_width, image_height, image_chunk);
        if ((hipError_t = hipDeviceSynchronize()) != hipSuccess) {
            std::wcout << std::endl << "[!] hipDeviceSynchronize(): hipError_t: " << hipError_t << std::endl;
            return -1;
        }

        hipMemcpy(&image[image_chunk], chunk_buffer_image, chunk_groups * threads * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }

    std::wcout << std::endl;

    return 0;
}